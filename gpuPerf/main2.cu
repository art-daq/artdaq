#include "hip/hip_runtime.h"

#include <iostream>
#include <stdexcept>
#include <vector>
#include <algorithm>
#include <cstdlib>

#include "alloc.hh"

using namespace std;

namespace cu
{
  // ----- will be useful when boost or c++11 is available
  class Stream
  {
  public:
    Stream() { hipStreamCreate(&s_); cerr<<"cuda strea made\n"; }
    ~Stream() { hipStreamDestroy(s_); cerr <<"cuda stream dtor\n";}

    operator hipStream_t&() { return s_; }

  private:
    hipStream_t s_;

    // really do not want this thing copied
    Stream(Stream const& c)
    { hipStreamCreate(&s_); cerr<<"cuda stream copy\n"; }
  };

  template<typename T>
  class DataDeviceAddr
  {
  public:
    explicit DataDeviceAddr(size_t num_elements):ptr_(0)
    {
      hipError_t err = hipMalloc(&ptr_,num_elements*sizeof(T));
      if(err!=hipSuccess) throw runtime_error("hipMalloc failed");
    }
    ~DataDeviceAddr() { hipFree(ptr_); }

    operator T*() { return (T*)ptr_; }
  private:
    void* ptr_;
    // no copy allowed
    DataDeviceAddr(DataDeviceAddr const&) { }
  };

  typedef std::vector<unsigned long, cu::allocator_host<unsigned long> > DataHostVec;

  class StreamVec
  {
  public:
    typedef std::vector<hipStream_t> Streams;

    explicit StreamVec(size_t count):strs_(count)
    {
      for(Streams::iterator i=strs_.begin(),e=strs_.end();
	  i!=e;++i)
	hipStreamCreate(&(*i));
    }
    ~StreamVec()
    {
      for(Streams::iterator i=strs_.begin(),e=strs_.end();
	  i!=e;++i)
	hipStreamDestroy((*i));
    }

    hipStream_t& operator[](int i) { return strs_[i]; }
  private:
    Streams strs_;
  };
};

// no good...
// typedef std::vector<cu::Stream> StreamVec;

struct GetRand {
  unsigned long operator()() const
  { long v = lrand48(); return (unsigned long)v; }
};

const unsigned long byte_count = 1ul<<30;

__global__ void kern_add(unsigned long* a, int size)
{
  *a += 1;
}

int main(int argc, char* argv[])
{
  if(argc<2)
    {
      cerr << "usage: " << argv[0] 
	   << " num_streams"
	   << "\n";
      return -1;
    }
  
  hipDeviceProp_t props; 
  hipGetDeviceProperties(&props,0);
  size_t max_threads_block = props.maxThreadsPerBlock;
  size_t max_threads_mp = props.maxThreadsPerMultiProcessor;

  size_t num_streams = atoi(argv[1]);
  size_t num_iters = argc>2?atoi(argv[2]):10;
  size_t num_elements = byte_count / sizeof(cu::DataHostVec::value_type);
  size_t elements_per = num_elements / num_streams;
  size_t bytes_per = elements_per * sizeof(cu::DataHostVec::value_type);

  cout << "allocating space for " << num_elements << " elements..." << endl;
  cu::StreamVec streams(num_streams);
  cout << "streams complete..." << endl;
  cu::DataHostVec h_in(num_elements);
  cout << "host in complete..." << endl;
  cu::DataHostVec h_out(num_elements);
  cout << "host out complete..." << endl;
  cu::DataDeviceAddr<unsigned long> d_mem(num_elements);
  cout << "device complete..." << endl;

  // sharedMemBytes - Dynamic shared-memory size per thread block in bytes
  int shared_mem = 0;
  // grid and block dimension can be specified in dim3 or int.
  // dim3 grid_dim(x,y,z);
  // dim3 block_dim(x,y,z);
  int grid_dim = 1;
  int block_dim = 512;
  int count_per_thread = byte_count / (grid_dim * block_dim * num_streams);

  // fill in_data with values
  cout << "filling data values..." << endl;
  generate(h_in.begin(),h_in.end(),GetRand());

  cout << "processing..." << endl;
  for(int iter=0;iter<num_iters;++iter)
    {
      int si = iter % num_streams;
      int di = si * elements_per;

      cout << "iter " << iter << " si=" << si << " di=" << di << endl;

      hipMemcpyAsync(d_mem+di, &h_in[di],
		      bytes_per, hipMemcpyHostToDevice, streams[si]);

      cout << "copy to device done... (" << &d_mem[di] << ")" << endl;

      kern_add<<<grid_dim, block_dim, shared_mem, streams[si]>>>
	(&d_mem[di], count_per_thread);

      cout << "kernel exec done..." << endl;

      hipMemcpyAsync(&h_out[di], &d_mem[di],
		      bytes_per, hipMemcpyDeviceToHost, streams[si]);

      cout << "copy to host done... (" << &d_mem[di] << ")" << endl;
    }

  hipDeviceSynchronize();
  cout << "ending" << endl;
  return 0;
}
