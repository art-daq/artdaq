#include "hip/hip_runtime.h"

#include <iostream>

using namespace std;

__global__ void add(int* a, int* b, int* c)
{
	*c = *a + *b;
}

ostream& operator<<(ostream& ost, hipDeviceProp_t const& p)
{
  ost << "major=" << p.major << "\n";
  ost << "minor=" << p.minor << "\n";
  ost << "MP count=" << p.multiProcessorCount << "\n";
  ost << "name=" << p.name << "\n";
  ost << "clock rate=" << p.clockRate << "\n";
  ost << "deviceOverlap=" << p.deviceOverlap << "\n";
  ost << "maxThreadsPerBlock=" << p.maxThreadsPerBlock << "\n";
  ost << "maxThreadsPerMP=" << p.maxThreadsPerMultiProcessor << "\n";
  ost << "warpSize=" << p.warpSize << "\n";
  // ost << "asyncEngineCount=" << p.asyncEngineCount << "\n";
  // ost << "concurrentKernels=" << p.concurrentKernels << "\n";
  ost << "integrated=" << p.integrated << "\n";
  return ost;
}

int main(void) 
{
  int a=1,b=2,c=0;
  int *pa=0,*pb=0,*pc=0;

  hipMalloc((void**)&pa,sizeof(int));
  hipMalloc((void**)&pb,sizeof(int));
  hipMalloc((void**)&pc,sizeof(int));

  hipMemcpy(pa,&a,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(pb,&b,sizeof(int),hipMemcpyHostToDevice);

  add<<<1,1>>>(pa,pb,pc);

  hipMemcpy(&c,pc,sizeof(int),hipMemcpyDeviceToHost);

  hipFree(pa);
  hipFree(pb);
  hipFree(pc);

  int dcount;
  hipGetDeviceCount(&dcount);

  hipDeviceProp_t prop0,prop1;
  hipGetDeviceProperties(&prop0,0);
  hipGetDeviceProperties(&prop1,0);

  cout << "c=" << c << endl;
  cout << "count=" << dcount << endl;
  cout << prop0 << "\n\n";
  cout << prop1 << "\n\n";

  return 0;
}
